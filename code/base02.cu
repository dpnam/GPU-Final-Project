
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

struct GpuTimer {
  hipEvent_t start;
  hipEvent_t stop;

  GpuTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  ~GpuTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void Start() {
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
  }

  void Stop() { hipEventRecord(stop, 0); }

  float Elapsed() {
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
  }
};

// Sequential radix sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
void sortByHost(const uint32_t *in, int n, uint32_t *out, int nBits) {
  int nBins = 1 << nBits; // 2^nBits
  int *hist = (int *)malloc(nBins * sizeof(int));
  int *histScan = (int *)malloc(nBins * sizeof(int));

  // In each counting sort, we sort data in "src" and write result to "dst"
  // Then, we swap these 2 pointers and go to the next counting sort
  // At first, we assign "src = in" and "dest = out"
  // However, the data pointed by "in" is read-only
  // --> we create a copy of this data and assign "src" to the address of this
  // copy
  uint32_t *src = (uint32_t *)malloc(n * sizeof(uint32_t));
  memcpy(src, in, n * sizeof(uint32_t));
  uint32_t *originalSrc = src; // Use originalSrc to free memory later
  uint32_t *dst = out;

  // Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
  // (Each digit consists of nBits bits)
  // In each loop, sort elements according to the current digit
  // (using STABLE counting sort)
  for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits) {
    // TODO: Compute "hist" of the current digit
    memset(hist, 0, nBins * sizeof(int));
    for (int i = 0; i < n; ++i) {
      int bin = (src[i] >> bit) & (nBins - 1);
      hist[bin]++;
    }

    // TODO: Scan "hist" (exclusively) and save the result to "histScan"
    histScan[0] = 0;
    for (int bin = 1; bin < nBins; ++bin) {
      histScan[bin] = histScan[bin - 1] + hist[bin - 1];
    }

    // TODO: From "histScan", scatter elements in "src" to correct locations in
    // "dst"
    for (int i = 0; i < n; ++i) {
      int bin = (src[i] >> bit) & (nBins - 1);
      dst[histScan[bin]] = src[i];
      histScan[bin]++;
    }

    // TODO: Swap "src" and "dst"
    uint32_t *temp = src;
    src = dst;
    dst = temp;
  }

  // TODO: Copy result to "out"
  if (src != out) {
    memcpy(out, src, n * sizeof(uint32_t));
  }

  // Free memories
  free(hist);
  free(histScan);
  free(originalSrc);
}

__global__ void computeHistKernel(uint32_t *in, int n, int *hist, int nBins,
                                  int bit) {
  // TODO
  extern __shared__ int s_hist[];
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  for (int s_i = threadIdx.x; s_i < nBins; s_i += blockDim.x) {
    s_hist[s_i] = 0;
  }
  __syncthreads();

  // Each block computes its local hist using atomic on SMEM
  if (i < n) {
    int bin = (in[i] >> bit) & (nBins - 1);
    atomicAdd(&s_hist[bin], 1);
  }

  __syncthreads();

  // Each block adds its local hist to global hist using atomic on GMEM
  for (int s_i = threadIdx.x; s_i < nBins; s_i += blockDim.x) {
    atomicAdd(&hist[s_i], s_hist[s_i]);
  }
}

__global__ void scanBlkKernel(int *in, int n, int *out, int *blkSums) {
  // TODO
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) {
    return;
  }

  extern __shared__ int s_in[];
  s_in[threadIdx.x] = in[i];
  __syncthreads();

  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    int strideVal;
    if (threadIdx.x >= stride) {
      strideVal = s_in[threadIdx.x - stride];
    }
    __syncthreads();

    if (threadIdx.x >= stride) {
      s_in[threadIdx.x] += strideVal;
    }
    __syncthreads();
  }

  if (blkSums && threadIdx.x == blockDim.x - 1) {
    blkSums[blockIdx.x] = s_in[threadIdx.x];
  }

  out[i] = s_in[threadIdx.x];
}

// TODO: You can define necessary functions here
__global__ void addBlkSums(int *in, int n, int *blkSums) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) {
    return;
  }

  in[i] += blkSums[blockIdx.x];
}

// (Partially) Parallel radix sort: implement parallel histogram and parallel
// scan in counting sort Assume: nBits (k in slides) in {1, 2, 4, 8, 16} Why
// "int * blockSizes"? Because we may want different block sizes for diffrent
// kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
void sortByDevice(const uint32_t *in, int n, uint32_t *out, int nBits,
                  int *blockSizes) {
  // TODO
  int nBins = 1 << nBits;

  dim3 histBlockSize(blockSizes[0]);
  dim3 histGridSize((n - 1) / histBlockSize.x + 1);
  size_t histSmemSize = nBins * sizeof(int);

  dim3 scanBlockSize(blockSizes[1]);
  int scanBlockCount = (nBins - 1) / scanBlockSize.x + 1;
  dim3 scanGridSize(scanBlockCount);
  size_t scanSmemSize = scanBlockSize.x * sizeof(int);

  uint32_t *d_in;
  int *d_hist;
  int *d_histScan;
  int *d_blkSums;

  CHECK(hipMalloc(&d_in, n * sizeof(uint32_t)));
  CHECK(hipMalloc(&d_hist, nBins * sizeof(int)));
  CHECK(hipMalloc(&d_histScan, nBins * sizeof(int)));
  CHECK(hipMalloc(&d_blkSums, scanBlockCount * sizeof(int)));

  uint32_t *src = (uint32_t *)malloc(n * sizeof(uint32_t));
  memcpy(src, in, n * sizeof(uint32_t));
  uint32_t *originalSrc = src; // Use originalSrc to free memory later
  uint32_t *dst = out;
  int *histScan = (int *)malloc(nBins * sizeof(int));

  int *blkSums = (int *)malloc(scanBlockCount * sizeof(int));

  for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits) {
    // Compute "hist" of the current digit
    CHECK(hipMemcpy(d_in, src, n * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECK(hipMemset(d_hist, 0, nBins * sizeof(int)));
    computeHistKernel<<<histGridSize, histBlockSize, histSmemSize>>>(
        d_in, n, d_hist, nBins, bit);
    CHECK(hipPeekAtLastError());

    // Scan "hist" (exclusively) and save the result to "histScan"
    scanBlkKernel<<<scanGridSize, scanBlockSize, scanSmemSize>>>(
        d_hist, nBins - 1, d_histScan + 1, d_blkSums);
    CHECK(hipPeekAtLastError());
    CHECK(hipMemcpy(blkSums, d_blkSums, scanBlockCount * sizeof(int),
                     hipMemcpyDeviceToHost));
    for (int i = 1; i < scanBlockCount; ++i) {
      blkSums[i] = blkSums[i - 1] + blkSums[i];
    }
    CHECK(hipMemcpy(d_blkSums, blkSums, (scanBlockCount - 1) * sizeof(int),
                     hipMemcpyHostToDevice));
    addBlkSums<<<scanGridSize, scanBlockSize>>>(
        d_histScan + scanBlockSize.x, nBins - scanBlockSize.x, d_blkSums);
    CHECK(hipPeekAtLastError());

    // From "histScan", scatter elements in "src" to correct locations in "dst"
    CHECK(hipMemcpy(histScan, d_histScan, nBins * sizeof(int),
                     hipMemcpyDeviceToHost));
    histScan[0] = 0;
    for (int i = 0; i < n; ++i) {
      int bin = (src[i] >> bit) & (nBins - 1);
      dst[histScan[bin]] = src[i];
      histScan[bin]++;
    }

    // Swap "src" and "dst"
    uint32_t *temp = src;
    src = dst;
    dst = temp;
  }

  // Copy result to "out"
  if (src != out) {
    memcpy(out, src, n * sizeof(uint32_t));
  }

  CHECK(hipFree(d_in));
  CHECK(hipFree(d_hist));
  CHECK(hipFree(d_histScan));
  CHECK(hipFree(d_blkSums));
  free(histScan);
  free(blkSums);
  free(originalSrc);
}

// Radix sort
void sort(const uint32_t *in, int n, uint32_t *out, int nBits,
          bool useDevice = false, int *blockSizes = NULL) {
  GpuTimer timer;
  timer.Start();

  if (useDevice == false) {
    printf("\nRadix sort by host\n");
    sortByHost(in, n, out, nBits);
  } else // use device
  {
    printf("\nRadix sort by device\n");
    sortByDevice(in, n, out, nBits, blockSizes);
  }

  timer.Stop();
  printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo() {
  hipDeviceProp_t devProv;
  CHECK(hipGetDeviceProperties(&devProv, 0));
  printf("**********GPU info**********\n");
  printf("Name: %s\n", devProv.name);
  printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
  printf("Num SMs: %d\n", devProv.multiProcessorCount);
  printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
  printf("Max num warps per SM: %d\n",
         devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
  printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
  printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
  printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
  printf("****************************\n");
}

void checkCorrectness(uint32_t *out, uint32_t *correctOut, int n) {
  for (int i = 0; i < n; i++) {
    if (out[i] != correctOut[i]) {
      printf("INCORRECT :(\n");
      return;
    }
  }
  printf("CORRECT :)\n");
}

void printArray(uint32_t *a, int n) {
  for (int i = 0; i < n; i++)
    printf("%i ", a[i]);
  printf("\n");
}

int main(int argc, char **argv) {
  // PRINT OUT DEVICE INFO
  printDeviceInfo();

  // SET UP INPUT SIZE
  int n = (1 << 24) + 1;
  // n = 10;
  printf("\nInput size: %d\n", n);

  // ALLOCATE MEMORIES
  size_t bytes = n * sizeof(uint32_t);
  uint32_t *in = (uint32_t *)malloc(bytes);
  uint32_t *out = (uint32_t *)malloc(bytes);        // Device result
  uint32_t *correctOut = (uint32_t *)malloc(bytes); // Host result

  // SET UP INPUT DATA
  for (int i = 0; i < n; i++)
    in[i] = rand();

  // SET UP NBITS
  int nBits = 4; // Default
  if (argc > 1)
    nBits = atoi(argv[1]);
  printf("\nNum bits per digit: %d\n", nBits);

  // DETERMINE BLOCK SIZES
  int blockSizes[2] = {512, 512}; // One for histogram, one for scan
  if (argc == 4) {
    blockSizes[0] = atoi(argv[2]);
    blockSizes[1] = atoi(argv[3]);
  }
  printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0],
         blockSizes[1]);

  // SORT BY HOST
  sort(in, n, correctOut, nBits);

  // SORT BY DEVICE
  sort(in, n, out, nBits, true, blockSizes);
  checkCorrectness(out, correctOut, n);

  // FREE MEMORIES
  free(in);
  free(out);
  free(correctOut);

  return EXIT_SUCCESS;
}
